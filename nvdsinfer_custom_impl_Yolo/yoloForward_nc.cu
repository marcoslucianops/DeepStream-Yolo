/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void gpuYoloLayer_nc(const float* input, float* output, const uint netWidth, const uint netHeight,
    const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses, const uint numBBoxes,
    const uint64_t lastInputSize, const float scaleXY, const float* anchors, const int* mask)
{
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes) {
    return;
  }

  const int numGridCells = gridSizeX * gridSizeY;
  const int bbindex = y_id * gridSizeX + x_id;

  const float alpha = scaleXY;
  const float beta = -0.5 * (scaleXY - 1);

  float xc = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)] * alpha + beta + x_id) * netWidth /
      gridSizeX;

  float yc = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)] * alpha + beta + y_id) * netHeight /
      gridSizeY;

  float w = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] * 2, 2) *
      anchors[mask[z_id] * 2];

  float h = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] * 2, 2) *
      anchors[mask[z_id] * 2 + 1];

  const float objectness = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  int count = numGridCells * z_id + bbindex + lastInputSize;

  output[count * 6 + 0] = xc - w * 0.5;
  output[count * 6 + 1] = yc - h * 0.5;
  output[count * 6 + 2] = xc + w * 0.5;
  output[count * 6 + 3] = yc + h * 0.5;
  output[count * 6 + 4] = maxProb * objectness;
  output[count * 6 + 5] = (float) maxIndex;
}

hipError_t cudaYoloLayer_nc(const void* input, void* output, const uint& batchSize, const uint64_t& inputSize,
    const uint64_t& outputSize, const uint64_t& lastInputSize, const uint& netWidth, const uint& netHeight,
    const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream);

hipError_t cudaYoloLayer_nc(const void* input, void* output, const uint& batchSize, const uint64_t& inputSize,
    const uint64_t& outputSize, const uint64_t& lastInputSize, const uint& netWidth, const uint& netHeight,
    const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream)
{
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1, (gridSizeY / threads_per_block.y) + 1,
      (numBBoxes / threads_per_block.z) + 1);

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuYoloLayer_nc<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float*> (input) + (batch * inputSize),
        reinterpret_cast<float*> (output) + (batch * 6 * outputSize),
        netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes, lastInputSize, scaleXY,
        reinterpret_cast<const float*> (anchors), reinterpret_cast<const int*> (mask));
  }
  return hipGetLastError();
}
